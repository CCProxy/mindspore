#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/hal/device/gpu_hash_table.h"

#if CUDA_VERSION > 11000
#include <cuco/dynamic_map.cuh>
#include <random>
#include <algorithm>

#include "plugin/device/gpu/hal/device/gpu_hash_table_kernel.cuh"
#include "utils/log_adapter.h"
#include "utils/convert_utils_base.h"
#include "plugin/device/gpu/hal/device/gpu_device_manager.h"

namespace mindspore {
namespace device {
namespace gpu {
#define CHECK_CUDA_RET(expression, message)                                                \
  {                                                                                        \
    hipError_t cuda_ret = (expression);                                                   \
    if (cuda_ret != hipSuccess) {                                                         \
      MS_LOG(ERROR) << "CUDA Error: " << message << " | Error Number: " << cuda_ret << " " \
                    << hipGetErrorString(cuda_ret);                                       \
    }                                                                                      \
  }

#define CHECK_CUDA_RET_WITH_RETURN_FALSE(expression, message)                              \
  {                                                                                        \
    hipError_t cuda_ret = (expression);                                                   \
    if (cuda_ret != hipSuccess) {                                                         \
      MS_LOG(ERROR) << "CUDA Error: " << message << " | Error Number: " << cuda_ret << " " \
                    << hipGetErrorString(cuda_ret);                                       \
      return false;                                                                        \
    }                                                                                      \
  }

#define ASSERT_EQUAL(lhs, rhs, message) \
  {                                     \
    if ((lhs) != (rhs)) {               \
      MS_LOG(ERROR) << message;         \
      return false;                     \
    }                                   \
  }

// The empty key, empty value(index) and erased key of CucoDynamicMap.
constexpr static int kEmptyKey = -1;
constexpr static int kEmptyValue = -1;
constexpr static int kErasedKey = -2;

template <typename Key, typename Value, typename Allocator>
using CucoDynamicMap = cuco::dynamic_map<Key, Value, cuda::thread_scope_device, Allocator>;

// CudaDynamicMap is a wrapper of cuco::dynamic_map, gpu_hash_table.h needs to be used by other cpp source files, in
// order for g++ to compile properly, the declaration of the cuco::dynamic_map type cannot appear in the header file
// gpu_hash_table.h, through the CudaDynamicMap type gpu_hash_ table.h pre-declaration to solve compilation problems.
template <typename Key, typename Value, typename Allocator>
struct CudaDynamicMap {
  CucoDynamicMap<Key, Value, Allocator> dynamic_map_;

  CudaDynamicMap(const Key &empty_key, const Value &empty_value, const Key &erased_key, const Allocator &alloc,
                 hipStream_t stream = 0)
      : dynamic_map_(kInitialCapacity, cuco::sentinel::empty_key<Key>{empty_key},
                     cuco::sentinel::empty_value<Value>{empty_value}, cuco::sentinel::erased_key<Key>{erased_key},
                     alloc, stream) {}

  ~CudaDynamicMap() = default;
};

template <typename Key, typename Value, typename Allocator>
GPUHashTable<Key, Value, Allocator>::GPUHashTable(int32_t value_dim, const std::string &initializer,
                                                  const Allocator &alloc)
    : value_dim_(value_dim), initializer_(initializer), default_value_(0), char_alloc_(alloc) {
  Initialize(alloc);
}

template <typename Key, typename Value, typename Allocator>
GPUHashTable<Key, Value, Allocator>::GPUHashTable(int32_t value_dim, const Value &default_value, const Allocator &alloc)
    : value_dim_(value_dim), initializer_(""), default_value_(default_value), char_alloc_(alloc) {
  Initialize(alloc);
}

template <typename Key, typename Value, typename Allocator>
GPUHashTable<Key, Value, Allocator>::~GPUHashTable() {
  Finalize();
}

template <typename Key, typename Value, typename Allocator>
void GPUHashTable<Key, Value, Allocator>::Initialize(const Allocator &alloc) {
  hipStream_t stream = reinterpret_cast<hipStream_t>(GPUDeviceManager::GetInstance().default_stream());
  cuda_dynamic_map_ = std::make_unique<CudaDynamicMap<Key, int32_t, Allocator>>(
    static_cast<Key>(kEmptyKey), kEmptyValue, static_cast<Key>(kErasedKey), alloc, stream);

  CudaAtomicSize host_init_atomic_size_t(0);
  CudaAtomicInt host_init_atomic_int(0);

  AllocateMemory(sizeof(CudaAtomicSize), &current_index_);
  AllocateMemory(sizeof(CudaAtomicInt), &erased_counter_);

  CHECK_CUDA_RET(
    hipMemcpyAsync(current_index_, &host_init_atomic_size_t, sizeof(CudaAtomicSize), hipMemcpyHostToDevice, stream),
    "hipMemcpyAsync");
  CHECK_CUDA_RET(
    hipMemcpyAsync(erased_counter_, &host_init_atomic_int, sizeof(CudaAtomicInt), hipMemcpyHostToDevice, stream),
    "hipMemcpyAsync");

  CHECK_CUDA_RET(hipStreamSynchronize(stream), "hipStreamSynchronize default cuda stream");

  CHECK_CUDA_RET(hipMallocManaged(&insert_success_number_, sizeof(CudaAtomicSize)), "hipMallocManaged");
}

template <typename Key, typename Value, typename Allocator>
void GPUHashTable<Key, Value, Allocator>::Finalize() {
  cuda_dynamic_map_ = nullptr;

  FreeMemory(current_index_);
  FreeMemory(erased_counter_);

  for (size_t i = 0; i < blocks_.size(); i++) {
    FreeMemory(blocks_[i]);
    FreeMemory(idle_flags_[i]);
  }

  if (blocks_ptr_) {
    FreeMemory(blocks_ptr_);
  }
  if (idle_flags_ptr_) {
    FreeMemory(idle_flags_ptr_);
  }

  if (random_gen_state_) {
    FreeMemory(random_gen_state_);
  }

  CHECK_CUDA_RET(hipFree(insert_success_number_), "hipFree");
}

template <typename Key, typename Value, typename Allocator>
template <typename T>
void GPUHashTable<Key, Value, Allocator>::AllocateMemory(size_t size, T **ptr) {
  MS_EXCEPTION_IF_NULL(ptr);
  *ptr = reinterpret_cast<T *>(std::allocator_traits<CharAllocatorType>::allocate(char_alloc_, size));
}

template <typename Key, typename Value, typename Allocator>
void GPUHashTable<Key, Value, Allocator>::FreeMemory(void *ptr) {
  std::allocator_traits<CharAllocatorType>::deallocate(char_alloc_, reinterpret_cast<char *>(ptr), 0);
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Find(const Key *keys, size_t key_num, Value *outputs, void *stream) {
  if (!initializer_.empty()) {
    return Find(keys, key_num, initializer_, outputs, stream);
  }
  return Find(keys, key_num, default_value_, outputs, stream);
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Find(const Key *keys, size_t key_num, const std::string &initializer,
                                               Value *outputs, void *stream) {
  MS_ERROR_IF_NULL(keys);
  MS_ERROR_IF_NULL(outputs);
  MS_ERROR_IF_NULL(stream);
  int *indices = nullptr;
  AllocateMemory(key_num * sizeof(int), &indices);
  MS_ERROR_IF_NULL(indices);
  Reserve(size_ + key_num, stream);

  // 1. Get all indices in blocks according to the keys.
  auto hip_stream = reinterpret_cast<hipStream_t>(stream);
  RETURN_IF_FALSE_WITH_LOG(GetIndicesByKeys(keys, key_num, true, indices, hip_stream), "Get indices by keys failed.");

  // 2. Insert default value according to initializer, initializer can be 'normal', 'zeros' or 'ones'.
  RETURN_IF_FALSE_WITH_LOG(InsertDefaultValueByInitializer(key_num, initializer, indices, hip_stream),
                           "Insert default value for miss keys failed.");

  // 3. Get all values by indices in blocks.
  size_t total_size = value_dim_ * key_num;
  GetValues<<<GET_BLOCKS(total_size), GET_THREADS, 0, hip_stream>>>(value_dim_, total_size, indices,
                                                                     elements_per_block_, blocks_ptr_, outputs);
  FreeMemory(indices);
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Find(const Key *keys, size_t key_num, const Value &default_value,
                                               Value *outputs, void *stream) {
  MS_ERROR_IF_NULL(keys);
  MS_ERROR_IF_NULL(outputs);
  MS_ERROR_IF_NULL(stream);
  int *indices = nullptr;
  AllocateMemory(key_num * sizeof(int), &indices);
  MS_ERROR_IF_NULL(indices);
  Reserve(size_ + key_num, stream);

  // 1. Get all indices in blocks according to the keys.
  auto hip_stream = reinterpret_cast<hipStream_t>(stream);
  RETURN_IF_FALSE_WITH_LOG(GetIndicesByKeys(keys, key_num, true, indices, hip_stream), "Get indices by keys failed.");

  // 2. Insert default value into map by specific value.
  InsertDefaultValue<<<GET_BLOCKS(key_num), GET_THREADS, 0, hip_stream>>>(
    value_dim_, key_num, indices, elements_per_block_, default_value, idle_flags_ptr_, blocks_ptr_);

  // 3. Get all values by indices in blocks.
  size_t total_size = value_dim_ * key_num;
  GetValues<<<GET_BLOCKS(total_size), GET_THREADS, 0, hip_stream>>>(value_dim_, total_size, indices,
                                                                     elements_per_block_, blocks_ptr_, outputs);
  FreeMemory(indices);
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Insert(const Key *keys, size_t key_num, const Value *value, void *stream) {
  MS_ERROR_IF_NULL(keys);
  MS_ERROR_IF_NULL(value);
  MS_ERROR_IF_NULL(stream);
  int *indices = nullptr;
  AllocateMemory(key_num * sizeof(int), &indices);
  MS_ERROR_IF_NULL(indices);
  Reserve(size_ + key_num, stream);

  // 1. Get all indices in blocks according to the keys.
  auto hip_stream = reinterpret_cast<hipStream_t>(stream);
  RETURN_IF_FALSE_WITH_LOG(GetIndicesByKeys(keys, key_num, true, indices, hip_stream), "Get indices by keys failed.");

  // 2. Insert values into map by indices in blocks.
  size_t total_insert_size = value_dim_ * key_num;
  InsertValues<<<GET_BLOCKS(total_insert_size), GET_THREADS, 0, hip_stream>>>(
    value_dim_, total_insert_size, indices, value, elements_per_block_, idle_flags_ptr_, blocks_ptr_);

  FreeMemory(indices);
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Erase(const Key *keys, size_t key_num, void *stream) {
  MS_ERROR_IF_NULL(keys);
  MS_ERROR_IF_NULL(stream);

  int *indices = nullptr;
  AllocateMemory(key_num * sizeof(int), &indices);
  MS_ERROR_IF_NULL(indices);

  // 1. Get all indices in blocks according to the key.
  auto hip_stream = reinterpret_cast<hipStream_t>(stream);
  RETURN_IF_FALSE_WITH_LOG(GetIndicesByKeys(keys, key_num, false, indices, hip_stream), "Get indices by keys failed.");

  // 2. Update idle status for erased slot.
  EraseElements<<<GET_BLOCKS(key_num), GET_THREADS, 0, hip_stream>>>(key_num, elements_per_block_, kEmptyValue,
                                                                      indices, idle_flags_ptr_);

  // 3. Erase all keys in dynamic map.
  MS_ERROR_IF_NULL(cuda_dynamic_map_);
  auto &dynamic_map = cuda_dynamic_map_->dynamic_map_;
  dynamic_map.erase(keys, keys + key_num, reinterpret_cast<hipStream_t>(stream));

  // 4. Update size.
  size_ = dynamic_map.get_size();

  FreeMemory(indices);
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Clear() {
  hipStream_t stream = reinterpret_cast<hipStream_t>(GPUDeviceManager::GetInstance().default_stream());
  // Need wait all task on stream finish.
  CHECK_CUDA_RET_WITH_RETURN_FALSE(hipStreamSynchronize(stream), "hipStreamSynchronize default cuda stream");

  size_ = 0;
  // 1. Reset cuda dynamic map.
  cuda_dynamic_map_ = std::make_unique<CudaDynamicMap<Key, int32_t, Allocator>>(
    static_cast<Key>(-1), -1, static_cast<Key>(-2), Allocator(), stream);

  CudaAtomicSize host_init_atomic_size_t(0);
  CudaAtomicInt host_init_atomic_int(0);

  // 2. Reset cuda atomic counter.
  CHECK_CUDA_RET_WITH_RETURN_FALSE(
    hipMemcpyAsync(current_index_, &host_init_atomic_size_t, sizeof(CudaAtomicSize), hipMemcpyHostToDevice, stream),
    "hipMemcpyAsync");
  CHECK_CUDA_RET_WITH_RETURN_FALSE(
    hipMemcpyAsync(erased_counter_, &host_init_atomic_int, sizeof(CudaAtomicInt), hipMemcpyHostToDevice, stream),
    "hipMemcpyAsync");

  // 3. Reset idle status.
  std::vector<int8_t> init_idle_flags(elements_per_block_, 1);
  for (size_t i = 0; i < idle_flags_.size(); i++) {
    CHECK_CUDA_RET_WITH_RETURN_FALSE(
      hipMemcpyAsync(idle_flags_[i], init_idle_flags.data(), init_idle_flags.size() * sizeof(bool),
                      hipMemcpyHostToDevice, stream),
      "hipMemcpyAsync");
  }

  CHECK_CUDA_RET_WITH_RETURN_FALSE(hipStreamSynchronize(stream), "hipStreamSynchronize default cuda stream");
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Reserve(size_t new_capacity, void *stream) {
  // There is sufficient space in hash table, need not to reserve.
  if (capacity() >= new_capacity) {
    return true;
  }
  MS_ERROR_IF_NULL(stream);
  auto hip_stream = reinterpret_cast<hipStream_t>(stream);

  // Allocate new block and idle flag until the capacity of hash table reaches desired capacity.
  size_t old_blocks_num = blocks_.size();
  std::vector<int8_t> init_idle_flags(elements_per_block_, 1);
  size_t remain_num = new_capacity - capacity();
  while (remain_num > 0) {
    // Allocate a new block.
    Value *new_block = nullptr;
    AllocateMemory(value_dim_ * elements_per_block_ * sizeof(Value), &new_block);
    MS_ERROR_IF_NULL(new_block);
    blocks_.push_back(new_block);

    // Allocate a new idle flag for new block.
    bool *new_block_idle_flag = nullptr;
    AllocateMemory(elements_per_block_ * sizeof(bool), &new_block_idle_flag);
    MS_ERROR_IF_NULL(new_block_idle_flag);
    idle_flags_.push_back(new_block_idle_flag);

    // Set initialized value for idle flag.
    CHECK_CUDA_RET_WITH_RETURN_FALSE(
      hipMemcpyAsync(new_block_idle_flag, init_idle_flags.data(), init_idle_flags.size() * sizeof(bool),
                      hipMemcpyHostToDevice, hip_stream),
      "hipMemcpyAsync");

    remain_num -= std::min(remain_num, elements_per_block_);
    capacity_ += elements_per_block_;
  }

  // Wait all task on the stream finish, because the blocks_ptr_ need to reallocate, there may be some kernels are using
  // the blocks_ptr_ and idle_flags_ptr_.
  CHECK_CUDA_RET_WITH_RETURN_FALSE(hipStreamSynchronize(hip_stream), "hipStreamSynchronize");

  return ResetBlockAndIdleFlag(hip_stream);
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::ResetBlockAndIdleFlag(hipStream_t hip_stream) {
  // Free old GPU memory for blocks_ptr_ and idle_flags_.
  if (blocks_ptr_) {
    FreeMemory(blocks_ptr_);
  }
  if (idle_flags_ptr_) {
    FreeMemory(idle_flags_ptr_);
  }

  size_t cur_blocks_num = blocks_.size();
  // Allocate new GPU memory for blocks_ptr_.
  Value *new_blocks_ptr = nullptr;
  AllocateMemory(cur_blocks_num * sizeof(Value *), &new_blocks_ptr);
  blocks_ptr_ = reinterpret_cast<Value **>(new_blocks_ptr);
  MS_ERROR_IF_NULL(blocks_ptr_);

  // Allocate new GPU memory for idle_flags_ptr_.
  bool *new_idle_flags_ptr = nullptr;
  AllocateMemory(cur_blocks_num * sizeof(bool *), &new_idle_flags_ptr);
  idle_flags_ptr_ = reinterpret_cast<bool **>(new_idle_flags_ptr);
  MS_ERROR_IF_NULL(idle_flags_ptr_);

  // Update the content for blocks pointer recorder and idle flags pointer recorder.
  CHECK_CUDA_RET_WITH_RETURN_FALSE(
    hipMemcpyAsync(blocks_ptr_, blocks_.data(), cur_blocks_num * sizeof(Value *), hipMemcpyHostToDevice, hip_stream),
    "hipMemcpyAsync");
  CHECK_CUDA_RET_WITH_RETURN_FALSE(hipMemcpyAsync(idle_flags_ptr_, idle_flags_.data(), cur_blocks_num * sizeof(bool *),
                                                   hipMemcpyHostToDevice, hip_stream),
                                   "hipMemcpyAsync");
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::GetKeysAndValues(Key *keys, Value *values, void *stream) {
  MS_ERROR_IF_NULL(keys);
  MS_ERROR_IF_NULL(values);
  MS_ERROR_IF_NULL(cuda_dynamic_map_);
  auto &dynamic_map = cuda_dynamic_map_->dynamic_map_;
  int *indices = nullptr;
  AllocateMemory(size_ * sizeof(int), &indices);
  MS_ERROR_IF_NULL(indices);

  // 1. Export all keys and indices from dynamic map.
  auto hip_stream = reinterpret_cast<hipStream_t>(stream);
  RETURN_IF_FALSE_WITH_LOG(dynamic_map.get_keys_values(keys, indices, hip_stream),
                           "Get keys and values from cuda dynamic map failed.");

  // 2. Get all values by indices in blocks.
  size_t total_size = value_dim_ * size_;
  GetValues<<<GET_BLOCKS(total_size), GET_THREADS, 0, hip_stream>>>(value_dim_, total_size, indices,
                                                                     elements_per_block_, blocks_ptr_, values);
  FreeMemory(indices);
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Import(const DataLenPair &input_data) {
  // 1. Store input tensor data until receiving kImportTensorNum(3) input tensor.
  // Really import input data to hash table when receive kImportTensorNum(3) input tensor.
  static std::vector<DataLenPair> input_data_list;
  if (input_data_list.size() < kImportTensorNum) {
    input_data_list.emplace_back(input_data);
  }
  if (input_data_list.size() != kImportTensorNum) {
    return true;
  }

  const auto &input_keys = input_data_list[0];
  const auto &input_values = input_data_list[1];
  void *host_keys = input_keys.first;
  void *host_values = input_values.first;
  MS_ERROR_IF_NULL(host_keys);
  MS_ERROR_IF_NULL(host_values);

  size_t keys_len = input_keys.second;
  size_t values_len = input_values.second;

  // 2. Allocate temp buffer to keys and values.
  Key *device_keys = nullptr;
  AllocateMemory(keys_len, &device_keys);
  MS_ERROR_IF_NULL(device_keys);

  Value *device_values = nullptr;
  AllocateMemory(values_len, &device_values);
  MS_ERROR_IF_NULL(device_values);

  // 3. Copy input keys and values to device.
  hipStream_t stream = reinterpret_cast<hipStream_t>(GPUDeviceManager::GetInstance().default_stream());
  CHECK_CUDA_RET_WITH_RETURN_FALSE(hipMemcpyAsync(device_keys, host_keys, keys_len, hipMemcpyHostToDevice, stream),
                                   "hipMemcpyAsync");
  CHECK_CUDA_RET_WITH_RETURN_FALSE(
    hipMemcpyAsync(device_values, host_values, values_len, hipMemcpyHostToDevice, stream), "hipMemcpyAsync");

  // 4. Insert input keys and values to hash table.
  RETURN_IF_FALSE_WITH_LOG(Insert(device_keys, keys_len / sizeof(Key), device_values, stream),
                           "Insert keys and values failed.");
  CHECK_CUDA_RET_WITH_RETURN_FALSE(hipStreamSynchronize(stream), "hipStreamSynchronize default cuda stream");

  // 5. Free temp buffer to keys and values.
  FreeMemory(device_keys);
  FreeMemory(device_values);

  input_data_list.clear();
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Export(const DataLenPair &keys, const DataLenPair &values,
                                                 const DataLenPair &status) {
  MS_ERROR_IF_NULL(keys.first);
  MS_ERROR_IF_NULL(values.first);
  MS_ERROR_IF_NULL(status.first);

  size_t keys_len = size_ * sizeof(Key);
  size_t values_len = size_ * value_dim_ * sizeof(Value);
  size_t status_len = size_ * sizeof(Status);
  // 1. Check length for output tensor.
  ASSERT_EQUAL(
    keys_len, keys.second,
    std::string("Need keys len[") + std::to_string(keys_len) + "], but got:[" + std::to_string(keys.second) + "].");
  ASSERT_EQUAL(values_len, values.second,
               std::string("Need values len[") + std::to_string(values_len) + "], but got:[" +
                 std::to_string(values.second) + "].");
  ASSERT_EQUAL(status_len, status.second,
               std::string("Need status len[") + std::to_string(status_len) + "], but got:[" +
                 std::to_string(status.second) + "].");

  // 2. Allocate temp buffer to keys, values and status.
  Key *device_keys = nullptr;
  Value *device_values = nullptr;
  Status *device_status = nullptr;
  AllocateMemory(keys_len, &device_keys);
  AllocateMemory(values_len, &device_values);
  AllocateMemory(status_len, &device_status);
  MS_ERROR_IF_NULL(device_keys);
  MS_ERROR_IF_NULL(device_values);
  MS_ERROR_IF_NULL(device_status);

  // 3. Export all keys and indices and store into temp buffer.
  hipStream_t stream = reinterpret_cast<hipStream_t>(GPUDeviceManager::GetInstance().default_stream());
  RETURN_IF_FALSE_WITH_LOG(GetKeysAndValues(device_keys, device_values, stream), "Get keys and values failed.");

  // Note: Get all status.
  // 4. Copy keys, values and status from device temp buffer to host.
  CHECK_CUDA_RET_WITH_RETURN_FALSE(hipMemcpyAsync(keys.first, device_keys, keys_len, hipMemcpyDeviceToHost, stream),
                                   "hipMemcpyAsync");
  CHECK_CUDA_RET_WITH_RETURN_FALSE(
    hipMemcpyAsync(values.first, device_values, values_len, hipMemcpyDeviceToHost, stream), "hipMemcpyAsync");
  CHECK_CUDA_RET_WITH_RETURN_FALSE(
    hipMemcpyAsync(status.first, device_status, status_len, hipMemcpyDeviceToHost, stream), "hipMemcpyAsync");
  CHECK_CUDA_RET_WITH_RETURN_FALSE(hipStreamSynchronize(stream), "hipStreamSynchronize default cuda stream");

  // 5. Free temp buffer to keys, values and status.
  FreeMemory(device_keys);
  FreeMemory(device_values);
  FreeMemory(device_status);

  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::GetIndicesByKeys(const Key *key, size_t key_num, bool insert_miss_key,
                                                           int32_t *indices, hipStream_t stream) {
  MS_ERROR_IF_NULL(key);
  MS_ERROR_IF_NULL(indices);
  MS_ERROR_IF_NULL(stream);
  MS_ERROR_IF_NULL(cuda_dynamic_map_);
  auto &dynamic_map = cuda_dynamic_map_->dynamic_map_;
  if (insert_miss_key) {
    dynamic_map.reserve(key_num + dynamic_map.get_size());
  }

  size_t submap_idx = 0;
  uint32_t device_id = GET_CTX_DEVICE_ID;
  size_t remaining_key_num = key_num;
  MS_ERROR_IF_NULL(insert_success_number_);

  while (remaining_key_num > 0) {
    auto &submap_ptr = dynamic_map.get_submaps()[submap_idx];
    MS_ERROR_IF_NULL(submap_ptr);
    // 1. Get reamaining capacity in current submap, max load faltor and min insert size need to be considered.
    size_t submap_remaining_capacity =
      submap_ptr->get_capacity() * dynamic_map.get_max_load_factor() - submap_ptr->get_size();
    if (submap_remaining_capacity < dynamic_map.get_min_insert_size()) {
      submap_idx++;
      continue;
    }

    *(insert_success_number_) = 0;
    CHECK_CUDA_RET_WITH_RETURN_FALSE(hipMemPrefetchAsync(insert_success_number_, sizeof(CudaAtomicSize), device_id),
                                     "hipMemPrefetchAsync");

    // 2. Get the key number could be handled by current submap.
    size_t item_num = std::min(submap_remaining_capacity, remaining_key_num);
    const uint32_t tile_size = kTileSize;
    const uint32_t block_size = kBlockSize;
    if (IntToUint(GET_THREADS) < block_size) {
      MS_LOG(ERROR) << "The max thread per block is less than: " << block_size << " of this GPU";
    }
    const uint32_t grid_size = IntToUint(CUDA_BLOCKS_CAL(device_id, tile_size * item_num, block_size));

    // 3. Transform all keys into indices in blocks. If the key exist in map already ,just return the index,
    // otherwise find a valid position in block.
    LookupIndices<block_size, tile_size, Key, typename CucoDynamicMap<Key, int32_t, Allocator>::mutable_view_type,
                  typename CucoDynamicMap<Key, int32_t, Allocator>::view_type><<<grid_size, block_size, 0, stream>>>(
      key, item_num, insert_miss_key, dynamic_map.get_submaps().size(), submap_idx, erased_slot_, erased_counter_,
      dynamic_map.get_submap_mutable_views().data().get(), dynamic_map.get_submap_views().data().get(),
      insert_success_number_, current_index_, indices);

    // 4. Update size for dynamic map and static submap.
    CHECK_CUDA_RET_WITH_RETURN_FALSE(hipStreamSynchronize(stream), "hipStreamSynchronize");
    size_t insert_success_num = insert_success_number_->load(cuda::std::memory_order_relaxed);
    dynamic_map.update_submap_size(submap_idx, submap_ptr->get_size() + insert_success_num);
    dynamic_map.update_size(dynamic_map.get_size() + insert_success_num);
    size_ += insert_success_num;

    indices += item_num;
    key += item_num;
    remaining_key_num -= item_num;
    submap_idx++;
  }
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::InsertDefaultValueByInitializer(size_t key_num,
                                                                          const std::string &initializer,
                                                                          const int *indices, hipStream_t stream) {
  MS_ERROR_IF_NULL(indices);
  MS_ERROR_IF_NULL(stream);
  if (initializer == kNormalDistribution) {
    // Normal distribution.
    RETURN_IF_FALSE_WITH_LOG(InitNormalDistRandomGenerator(stream),
                             "Initialize normal distribution random generator failed.");
    Value mean = static_cast<Value>(0);
    Value stddev = static_cast<Value>(0.01);

    InsertNormalDistRandomValue<<<random_gen_block_count_, random_gen_threads_per_block_, 0, stream>>>(
      value_dim_, key_num, indices, elements_per_block_, mean, stddev, random_gen_state_, idle_flags_ptr_, blocks_ptr_);
  } else if (initializer == kOnesDistribution) {
    // One distribution.
    InsertDefaultValue<<<GET_BLOCKS(key_num), GET_THREADS, 0, stream>>>(
      value_dim_, key_num, indices, elements_per_block_, static_cast<Value>(1.0), idle_flags_ptr_, blocks_ptr_);
  } else if (initializer == kZerosDistribution) {
    // Zero distribution.
    InsertDefaultValue<<<GET_BLOCKS(key_num), GET_THREADS, 0, stream>>>(
      value_dim_, key_num, indices, elements_per_block_, static_cast<Value>(0), idle_flags_ptr_, blocks_ptr_);
  } else {
    MS_LOG(ERROR) << "Unsupported initializer: " << initializer;
    return false;
  }

  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::InitNormalDistRandomGenerator(hipStream_t stream) {
  MS_ERROR_IF_NULL(stream);
  if (random_gen_init_.load()) {
    return true;
  }

  // 1. Allocate memory for all random generator states.
  auto total_random_state_num = random_gen_threads_per_block_ * random_gen_block_count_;
  AllocateMemory(IntToSize(total_random_state_num) * sizeof(hiprandStatePhilox4_32_10_t), &random_gen_state_);
  MS_ERROR_IF_NULL(random_gen_state_);

  // 2. Initialize normal distribution random generator states.
  std::random_device rd;
  uint32_t seed = rd();
  InitNormalDisRandomGen<<<random_gen_block_count_, random_gen_threads_per_block_, 0, stream>>>(seed,
                                                                                                random_gen_state_);

  random_gen_init_ = true;
  return true;
}

template class GPUHashTable<int32_t, float>;
template class GPUHashTable<int64_t, float>;
}  // namespace gpu
}  // namespace device
}  // namespace mindspore
#endif
