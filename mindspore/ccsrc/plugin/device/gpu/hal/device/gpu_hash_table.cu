#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#if CUDA_VERSION > 11000
#include "plugin/device/gpu/hal/device/gpu_hash_table.h"

#include <cuco/dynamic_map.cuh>
#include <random>
#include <algorithm>

#include "plugin/device/gpu/hal/device/gpu_hash_table_kernel.cuh"
#include "utils/log_adapter.h"
#include "utils/convert_utils_base.h"
#include "plugin/device/gpu/hal/device/gpu_device_manager.h"

namespace mindspore {
namespace device {
namespace gpu {
#define CHECK_CUDA_RET(expression, message)                                                \
  {                                                                                        \
    hipError_t cuda_ret = (expression);                                                   \
    if (cuda_ret != hipSuccess) {                                                         \
      MS_LOG(ERROR) << "CUDA Error: " << message << " | Error Number: " << cuda_ret << " " \
                    << hipGetErrorString(cuda_ret);                                       \
    }                                                                                      \
  }

#define CHECK_CUDA_RET_WITH_RETURN_FALSE(expression, message)                              \
  {                                                                                        \
    hipError_t cuda_ret = (expression);                                                   \
    if (cuda_ret != hipSuccess) {                                                         \
      MS_LOG(ERROR) << "CUDA Error: " << message << " | Error Number: " << cuda_ret << " " \
                    << hipGetErrorString(cuda_ret);                                       \
      return false;                                                                        \
    }                                                                                      \
  }

#define ASSERT_EQUAL(lhs, rhs, message) \
  {                                     \
    if ((lhs) != (rhs)) {               \
      MS_LOG(ERROR) << message;         \
      return false;                     \
    }                                   \
  }

template <typename Key, typename Value, typename Allocator>
using CucoDynamicMap = cuco::dynamic_map<Key, Value, cuda::thread_scope_device, Allocator>;

// CudaDynamicMap is a wrapper of cuco::dynamic_map, gpu_hash_table.h needs to be used by other cpp source files, in
// order for g++ to compile properly, the declaration of the cuco::dynamic_map type cannot appear in the header file
// gpu_hash_table.h, through the CudaDynamicMap type gpu_hash_ table.h pre-declaration to solve compilation problems.
template <typename Key, typename Value, typename Allocator>
struct CudaDynamicMap {
  CucoDynamicMap<Key, Value, Allocator> dynamic_map_;

  CudaDynamicMap(const Key &empty_key, const Value &empty_value, const Key &erased_key, const Allocator &alloc,
                 hipStream_t stream = 0)
      : dynamic_map_(kInitialCapacity, cuco::sentinel::empty_key<Key>{empty_key},
                     cuco::sentinel::empty_value<Value>{empty_value}, cuco::sentinel::erased_key<Key>{erased_key},
                     alloc, stream) {}

  ~CudaDynamicMap() = default;
};

template <typename Key, typename Value, typename Allocator>
GPUHashTable<Key, Value, Allocator>::GPUHashTable(int32_t value_dim, const std::string &initializer,
                                                  const Allocator &alloc)
    : value_dim_(value_dim), initializer_(initializer), default_value_(0), index_alloc_(alloc), char_alloc_(alloc) {
  hipStream_t stream = reinterpret_cast<hipStream_t>(GPUDeviceManager::GetInstance().default_stream());
  cuda_dynamic_map_ = std::make_unique<CudaDynamicMap<Key, int32_t, Allocator>>(static_cast<Key>(-1), -1,
                                                                                static_cast<Key>(-2), alloc, stream);
  CHECK_CUDA_RET(hipStreamSynchronize(stream), "hipStreamSynchronize default cuda stream");

  CHECK_CUDA_RET(
    hipMallocManaged(&insert_success_number_, sizeof(cuda::atomic<std::size_t, cuda::thread_scope_device>)),
    "hipMallocManaged");
}

template <typename Key, typename Value, typename Allocator>
GPUHashTable<Key, Value, Allocator>::GPUHashTable(int32_t value_dim, const Value &default_value, const Allocator &alloc)
    : value_dim_(value_dim), initializer_(""), default_value_(default_value), index_alloc_(alloc), char_alloc_(alloc) {
  hipStream_t stream = reinterpret_cast<hipStream_t>(GPUDeviceManager::GetInstance().default_stream());
  cuda_dynamic_map_ = std::make_unique<CudaDynamicMap<Key, int32_t, Allocator>>(static_cast<Key>(-1), -1,
                                                                                static_cast<Key>(-2), alloc, stream);
  CHECK_CUDA_RET(hipStreamSynchronize(stream), "hipStreamSynchronize default cuda stream");

  CHECK_CUDA_RET(
    hipMallocManaged(&insert_success_number_, sizeof(cuda::atomic<std::size_t, cuda::thread_scope_device>)),
    "hipMallocManaged");
}

template <typename Key, typename Value, typename Allocator>
GPUHashTable<Key, Value, Allocator>::~GPUHashTable() {
  CHECK_CUDA_RET(hipFree(insert_success_number_), "hipFree");
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Find(const Key *keys, size_t key_num, Value *outputs, void *stream) {
  if (!initializer_.empty()) {
    return Find(keys, key_num, initializer_, outputs, stream);
  }
  return Find(keys, key_num, default_value_, outputs, stream);
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Find(const Key *key, size_t key_num, const std::string &initializer,
                                               Value *outputs, void *stream) {
  MS_ERROR_IF_NULL(key);
  MS_ERROR_IF_NULL(outputs);
  MS_ERROR_IF_NULL(stream);
  int *indices = std::allocator_traits<IndexAllocatorType>::allocate(index_alloc_, key_num);
  MS_ERROR_IF_NULL(indices);

  // 1. Get all indices in blocks according to the key.
  auto hip_stream = reinterpret_cast<hipStream_t>(stream);
  RETURN_IF_FALSE_WITH_LOG(GetIndicesByKeys(key, key_num, true, indices, hip_stream), "Get indices by keys failed.");

  Reserve(size_ + key_num);
  // 2. Insert default value according to initializer, initializer can be 'normal', 'zeros' or 'ones'.
  RETURN_IF_FALSE_WITH_LOG(InsertDefaultValueByInitializer(key_num, initializer, indices, hip_stream),
                           "Insert default value for miss keys failed.");

  // 3. Get all values by indices in blocks.
  size_t total_size = value_dim_ * key_num;
  GetValues<<<GET_BLOCKS(total_size), GET_THREADS, 0, hip_stream>>>(value_dim_, total_size, indices,
                                                                     elements_per_block_, blocks_ptr_, outputs);
  std::allocator_traits<IndexAllocatorType>::deallocate(index_alloc_, indices, key_num);
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Find(const Key *key, size_t key_num, const Value &default_value,
                                               Value *outputs, void *stream) {
  MS_ERROR_IF_NULL(key);
  MS_ERROR_IF_NULL(outputs);
  MS_ERROR_IF_NULL(stream);
  int *indices = std::allocator_traits<IndexAllocatorType>::allocate(index_alloc_, key_num);
  MS_ERROR_IF_NULL(indices);

  // 1. Get all indices in blocks according to the key.
  auto hip_stream = reinterpret_cast<hipStream_t>(stream);
  RETURN_IF_FALSE_WITH_LOG(GetIndicesByKeys(key, key_num, true, indices, hip_stream), "Get indices by keys failed.");

  Reserve(size_ + key_num);
  // 2. Insert default value into map by specific value.
  InsertDefaultValue<<<GET_BLOCKS(key_num), GET_THREADS, 0, hip_stream>>>(
    value_dim_, key_num, indices, elements_per_block_, default_value, idle_flags_ptr_, blocks_ptr_);

  // 3. Get all values by indices in blocks.
  size_t total_size = value_dim_ * key_num;
  GetValues<<<GET_BLOCKS(total_size), GET_THREADS, 0, hip_stream>>>(value_dim_, total_size, indices,
                                                                     elements_per_block_, blocks_ptr_, outputs);
  std::allocator_traits<IndexAllocatorType>::deallocate(index_alloc_, indices, key_num);
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Insert(const Key *key, size_t key_num, const Value *value, void *stream) {
  MS_ERROR_IF_NULL(key);
  MS_ERROR_IF_NULL(value);
  MS_ERROR_IF_NULL(stream);
  int *indices = std::allocator_traits<IndexAllocatorType>::allocate(index_alloc_, key_num);
  MS_ERROR_IF_NULL(indices);

  // 1. Get all indices in blocks according to the key.
  auto hip_stream = reinterpret_cast<hipStream_t>(stream);
  RETURN_IF_FALSE_WITH_LOG(GetIndicesByKeys(key, key_num, true, indices, hip_stream), "Get indices by keys failed.");

  Reserve(size_ + key_num);
  // 2. Insert values into map by indices in blocks.
  size_t total_insert_size = value_dim_ * key_num;
  InsertValues<<<GET_BLOCKS(total_insert_size), GET_THREADS, 0, hip_stream>>>(
    value_dim_, total_insert_size, indices, value, elements_per_block_, idle_flags_ptr_, blocks_ptr_);

  std::allocator_traits<IndexAllocatorType>::deallocate(index_alloc_, indices, key_num);
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Erase(const Key *keys, size_t key_num, void *stream) {
  MS_ERROR_IF_NULL(keys);
  MS_ERROR_IF_NULL(stream);
  MS_ERROR_IF_NULL(cuda_dynamic_map_);
  auto &dynamic_map = cuda_dynamic_map_->dynamic_map_;
  dynamic_map.erase(keys, keys + key_num, reinterpret_cast<hipStream_t>(stream));
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::GetKeysAndValues(Key *keys, Value *values, void *stream) {
  MS_ERROR_IF_NULL(keys);
  MS_ERROR_IF_NULL(values);
  MS_ERROR_IF_NULL(cuda_dynamic_map_);
  auto &dynamic_map = cuda_dynamic_map_->dynamic_map_;
  int *indices = std::allocator_traits<IndexAllocatorType>::allocate(index_alloc_, size_);
  MS_ERROR_IF_NULL(indices);

  // 1. Export all keys and indices from dynamic map.
  auto hip_stream = reinterpret_cast<hipStream_t>(stream);
  RETURN_IF_FALSE_WITH_LOG(dynamic_map.get_keys_values(keys, indices, hip_stream),
                           "Get keys and values from cuda dynamic map failed.");

  // 2. Get all values by indices in blocks.
  size_t total_size = value_dim_ * size_;
  GetValues<<<GET_BLOCKS(total_size), GET_THREADS, 0, hip_stream>>>(value_dim_, total_size, indices,
                                                                     elements_per_block_, blocks_ptr_, values);
  std::allocator_traits<IndexAllocatorType>::deallocate(index_alloc_, indices, size_);
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Import(const DataLenPair &input_data) {
  // 1. Store input tensor data until receiving kImportTensorNum(3) input tensor.
  // Really import input data to hash table when receive kImportTensorNum(3) input tensor.
  static std::vector<DataLenPair> input_data_list;
  if (input_data_list.size() < kImportTensorNum) {
    input_data_list.emplace_back(input_data);
  }
  if (input_data_list.size() != kImportTensorNum) {
    return true;
  }

  const auto &input_keys = input_data_list[0];
  const auto &input_values = input_data_list[1];
  void *host_keys = input_keys.first;
  void *host_values = input_values.first;
  MS_ERROR_IF_NULL(host_keys);
  MS_ERROR_IF_NULL(host_values);

  size_t keys_len = input_keys.second;
  size_t values_len = input_values.second;

  // 2. Allocate temp buffer to keys and values.
  char *device_keys = std::allocator_traits<CharAllocatorType>::allocate(char_alloc_, keys_len);
  char *device_values = std::allocator_traits<CharAllocatorType>::allocate(char_alloc_, values_len);
  MS_ERROR_IF_NULL(device_keys);
  MS_ERROR_IF_NULL(device_values);

  // 3. Copy input keys and values to device.
  hipStream_t stream = reinterpret_cast<hipStream_t>(GPUDeviceManager::GetInstance().default_stream());
  CHECK_CUDA_RET_WITH_RETURN_FALSE(hipMemcpyAsync(device_keys, host_keys, keys_len, hipMemcpyHostToDevice, stream),
                                   "hipMemcpyAsync");
  CHECK_CUDA_RET_WITH_RETURN_FALSE(
    hipMemcpyAsync(device_values, host_values, values_len, hipMemcpyHostToDevice, stream), "hipMemcpyAsync");

  // 4. Insert input keys and values to hash table.
  RETURN_IF_FALSE_WITH_LOG(Insert(reinterpret_cast<Key *>(device_keys), keys_len / sizeof(Key),
                                  reinterpret_cast<Value *>(device_values), stream),
                           "Insert keys and values failed.");
  CHECK_CUDA_RET_WITH_RETURN_FALSE(hipStreamSynchronize(stream), "hipStreamSynchronize default cuda stream");

  // 5. Free temp buffer to keys and values.
  std::allocator_traits<CharAllocatorType>::deallocate(char_alloc_, device_keys, keys_len);
  std::allocator_traits<CharAllocatorType>::deallocate(char_alloc_, device_values, values_len);
  input_data_list.clear();

  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::Export(const DataLenPair &keys, const DataLenPair &values,
                                                 const DataLenPair &status) {
  MS_ERROR_IF_NULL(keys.first);
  MS_ERROR_IF_NULL(values.first);
  MS_ERROR_IF_NULL(status.first);

  size_t keys_len = size_ * sizeof(Key);
  size_t values_len = size_ * value_dim_ * sizeof(Value);
  size_t status_len = size_ * sizeof(Status);
  // 1. Check length for output tensor.
  ASSERT_EQUAL(
    keys_len, keys.second,
    std::string("Need keys len[") + std::to_string(keys_len) + "], but got:[" + std::to_string(keys.second) + "].");
  ASSERT_EQUAL(values_len, values.second,
               std::string("Need values len[") + std::to_string(values_len) + "], but got:[" +
                 std::to_string(values.second) + "].");
  ASSERT_EQUAL(status_len, status.second,
               std::string("Need status len[") + std::to_string(status_len) + "], but got:[" +
                 std::to_string(status.second) + "].");

  // 2. Allocate temp buffer to keys, values and status.
  char *device_keys = std::allocator_traits<CharAllocatorType>::allocate(char_alloc_, keys_len);
  char *device_values = std::allocator_traits<CharAllocatorType>::allocate(char_alloc_, values_len);
  char *device_status = std::allocator_traits<CharAllocatorType>::allocate(char_alloc_, status_len);
  MS_ERROR_IF_NULL(device_keys);
  MS_ERROR_IF_NULL(device_values);
  MS_ERROR_IF_NULL(device_status);

  // 3. Export all keys and indices and store into temp buffer.
  hipStream_t stream = reinterpret_cast<hipStream_t>(GPUDeviceManager::GetInstance().default_stream());
  RETURN_IF_FALSE_WITH_LOG(
    GetKeysAndValues(reinterpret_cast<Key *>(device_keys), reinterpret_cast<Value *>(device_values), stream),
    "Get keys and values failed.");

  // Note: Get all status.
  // 4. Copy keys, values and status from device temp buffer to host.
  CHECK_CUDA_RET_WITH_RETURN_FALSE(hipMemcpyAsync(keys.first, device_keys, keys_len, hipMemcpyDeviceToHost, stream),
                                   "hipMemcpyAsync");
  CHECK_CUDA_RET_WITH_RETURN_FALSE(
    hipMemcpyAsync(values.first, device_values, values_len, hipMemcpyDeviceToHost, stream), "hipMemcpyAsync");
  CHECK_CUDA_RET_WITH_RETURN_FALSE(
    hipMemcpyAsync(status.first, device_status, status_len, hipMemcpyDeviceToHost, stream), "hipMemcpyAsync");
  CHECK_CUDA_RET_WITH_RETURN_FALSE(hipStreamSynchronize(stream), "hipStreamSynchronize default cuda stream");

  // 5. Free temp buffer to keys, values and status.
  std::allocator_traits<CharAllocatorType>::deallocate(char_alloc_, device_keys, keys_len);
  std::allocator_traits<CharAllocatorType>::deallocate(char_alloc_, device_values, values_len);
  std::allocator_traits<CharAllocatorType>::deallocate(char_alloc_, device_status, status_len);

  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::GetIndicesByKeys(const Key *key, size_t key_num, bool insert_miss_key,
                                                           int32_t *indices, hipStream_t stream) {
  MS_ERROR_IF_NULL(key);
  MS_ERROR_IF_NULL(indices);
  MS_ERROR_IF_NULL(stream);
  MS_ERROR_IF_NULL(cuda_dynamic_map_);
  auto &dynamic_map = cuda_dynamic_map_->dynamic_map_;
  dynamic_map.reserve(key_num + dynamic_map.get_size());
  size_t submap_idx = 0;
  uint32_t device_id = GET_CTX_DEVICE_ID;
  size_t remaining_key_num = key_num;
  MS_ERROR_IF_NULL(insert_success_number_);

  while (remaining_key_num > 0) {
    auto &submap_ptr = dynamic_map.get_submaps()[submap_idx];
    MS_ERROR_IF_NULL(submap_ptr);
    // 1. Get reamaining capacity in current submap, max load faltor and min insert size need to be considered.
    size_t submap_remaining_capacity =
      submap_ptr->get_capacity() * dynamic_map.get_max_load_factor() - submap_ptr->get_size();
    if (submap_remaining_capacity < dynamic_map.get_min_insert_size()) {
      submap_idx++;
      continue;
    }

    *(insert_success_number_) = 0;
    CHECK_CUDA_RET_WITH_RETURN_FALSE(hipMemPrefetchAsync(insert_success_number_, sizeof(CudaAtomicSize), device_id),
                                     "hipMemPrefetchAsync");

    // 2. Get the key number could be handled by current submap.
    size_t item_num = std::min(submap_remaining_capacity, remaining_key_num);
    const uint32_t tile_size = kTileSize;
    const uint32_t block_size = kBlockSize;
    if (IntToUint(GET_THREADS) < block_size) {
      MS_LOG(ERROR) << "The max thread per block is less than: " << block_size << " of this GPU";
    }
    const uint32_t grid_size = IntToUint(CUDA_BLOCKS_CAL(device_id, tile_size * item_num, block_size));

    // 3. Transform all keys into indices in blocks. If the key exist in map already ,just return the index,
    // otherwise find a valid position in block.
    LookupIndices<block_size, tile_size, Key, typename CucoDynamicMap<Key, int32_t, Allocator>::mutable_view_type,
                  typename CucoDynamicMap<Key, int32_t, Allocator>::view_type><<<grid_size, block_size, 0, stream>>>(
      key, item_num, insert_miss_key, dynamic_map.get_submaps().size(), submap_idx, idel_slot_, idle_index_,
      dynamic_map.get_submap_mutable_views().data().get(), dynamic_map.get_submap_views().data().get(),
      insert_success_number_, current_index_, indices);

    // 4. Update size for dynamic map and static submap.
    CHECK_CUDA_RET_WITH_RETURN_FALSE(hipStreamSynchronize(stream), "hipStreamSynchronize");
    size_t insert_success_num = insert_success_number_->load(cuda::std::memory_order_relaxed);
    dynamic_map.update_submap_size(submap_idx, submap_ptr->get_size() + insert_success_num);
    dynamic_map.update_size(dynamic_map.get_size() + insert_success_num);
    size_ += insert_success_num;

    indices += item_num;
    key += item_num;
    remaining_key_num -= item_num;
    submap_idx++;
  }
  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::InsertDefaultValueByInitializer(size_t key_num,
                                                                          const std::string &initializer,
                                                                          const int *indices, hipStream_t stream) {
  MS_ERROR_IF_NULL(indices);
  MS_ERROR_IF_NULL(stream);
  if (initializer == kNormalDistribution) {
    // Normal distribution.
    RETURN_IF_FALSE_WITH_LOG(InitNormalDistRandomGenerator(stream),
                             "Initialize normal distribution random generator failed.");
    Value mean = static_cast<Value>(0);
    Value stddev = static_cast<Value>(0.01);

    InsertNormalDistRandomValue<<<random_gen_block_count_, random_gen_threads_per_block_, 0, stream>>>(
      value_dim_, key_num, indices, elements_per_block_, mean, stddev, random_gen_state_, idle_flags_ptr_, blocks_ptr_);
  } else if (initializer == kOnesDistribution) {
    // One distribution.
    InsertDefaultValue<<<GET_BLOCKS(key_num), GET_THREADS, 0, stream>>>(
      value_dim_, key_num, indices, elements_per_block_, static_cast<Value>(1.0), idle_flags_ptr_, blocks_ptr_);
  } else if (initializer == kZerosDistribution) {
    // Zero distribution.
    InsertDefaultValue<<<GET_BLOCKS(key_num), GET_THREADS, 0, stream>>>(
      value_dim_, key_num, indices, elements_per_block_, static_cast<Value>(0), idle_flags_ptr_, blocks_ptr_);
  } else {
    MS_LOG(ERROR) << "Unsupported initializer: " << initializer;
    return false;
  }

  return true;
}

template <typename Key, typename Value, typename Allocator>
bool GPUHashTable<Key, Value, Allocator>::InitNormalDistRandomGenerator(hipStream_t stream) {
  MS_ERROR_IF_NULL(stream);
  if (random_gen_init_.load()) {
    return true;
  }

  // 1. Allocate memory for all random generator states.
  auto total_random_state_num = random_gen_threads_per_block_ * random_gen_block_count_;
  random_gen_state_ = reinterpret_cast<hiprandStatePhilox4_32_10_t *>(std::allocator_traits<CharAllocatorType>::allocate(
    char_alloc_, IntToSize(total_random_state_num) * sizeof(hiprandStatePhilox4_32_10_t)));
  MS_ERROR_IF_NULL(random_gen_state_);

  // 2. Initialize normal distribution random generator states.
  std::random_device rd;
  uint32_t seed = rd();
  InitNormalDisRandomGen<<<random_gen_block_count_, random_gen_threads_per_block_, 0, stream>>>(seed,
                                                                                                random_gen_state_);

  random_gen_init_ = true;
  return true;
}

template class GPUHashTable<int32_t, float>;
template class GPUHashTable<int64_t, float>;
}  // namespace gpu
}  // namespace device
}  // namespace mindspore
#endif
