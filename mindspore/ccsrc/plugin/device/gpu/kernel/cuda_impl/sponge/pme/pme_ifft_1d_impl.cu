/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "plugin/device/gpu/kernel/cuda_impl/sponge/pme/pme_ifft_1d_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/sponge/pme/pme_common.cuh"

template <typename T>
void PMEIFFT1D(int Nfft, Complex<T> *input_tensor, Complex<T> *output_tensor, const hipfftHandle &FFT_plan_c2c,
            hipStream_t stream) {
  hipfftExecC2C(FFT_plan_c2c, reinterpret_cast<hipfftComplex *>(input_tensor),
              reinterpret_cast<hipfftComplex *>(output_tensor), HIPFFT_BACKWARD);
  return;
}

template void PMEIFFT1D<float>(int Nfft, Complex<float> *input_tensor, Complex<float> *output_tensor,
                           const hipfftHandle &FFT_plan_c2c, hipStream_t stream);
