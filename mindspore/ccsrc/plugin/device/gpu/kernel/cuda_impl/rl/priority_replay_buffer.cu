#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/rl/priority_replay_buffer.cuh"
#include <limits>
#include <algorithm>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

constexpr float kInitPriority = 1.0;
constexpr size_t kRootIdx = 1;
constexpr size_t kNumSubNode = 2;
constexpr size_t kMaxThreadPerBlock = 128;

__global__ void SumTreeInitKernel(SumTree *tree, float *max_priority, size_t size) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += gridDim.x * blockDim.x) {
    tree[i].sum = 0;
    tree[i].min = std::numeric_limits<float>::max();

    if (i == 0) {
      *max_priority = kInitPriority;
    }
  }
}

__global__ void InitRandStateKernel(uint64_t seed, hiprandState *state) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, tid, 0, &(state[tid]));
}

__forceinline__ __device__ void SumTreeInsert(SumTree *tree, size_t idx, float priority) {
  tree[idx].sum = priority;
  tree[idx].min = priority;

  size_t parent = idx >> 1;
  while (parent >= kRootIdx) {
    size_t left_child = parent << 1;
    // TODO(WilfChen): Different thead may rewrite same node parallel since indices maybe repeat.
    tree[parent].sum = tree[left_child].sum + tree[left_child + 1].sum;
    tree[parent].min = min(tree[left_child].min, tree[left_child + 1].min);
    parent >>= 1;
  }
}

__global__ void SumTreePushKernel(SumTree *tree, float alpha, size_t idx, float *priority, float *max_priority) {
  float prio;
  if (!priority) {
    prio = powf(*max_priority, alpha);
  } else {
    *max_priority = max(*max_priority, *priority);
    prio = powf(*priority, alpha);
  }

  SumTreeInsert(tree, idx, prio);
}

__forceinline__ __device__ size_t GetPrefixSumIdx(SumTree *tree, size_t capacity, float prefix_sum) {
  size_t idx = kRootIdx;
  while (idx < capacity) {
    const float &left_priority = tree[kNumSubNode * idx].sum;
    if (prefix_sum <= left_priority) {
      idx = kNumSubNode * idx;
    } else {
      prefix_sum -= left_priority;
      idx = kNumSubNode * idx + 1;
    }
  }
  return idx - capacity;
}

__global__ void SumTreeSampleKernel(SumTree *tree, hiprandState *state, size_t capacity, float *beta, size_t batch_size,
                                    size_t *indices, float *weights) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < batch_size; i += gridDim.x * blockDim.x) {
    size_t segment_len = tree[kRootIdx].sum / batch_size;
    float prefix_sum = (hiprand_uniform(&state[i]) + i) * segment_len;
    size_t idx = GetPrefixSumIdx(tree, capacity, prefix_sum);
    indices[i] = idx;
    weights[i] = powf((tree[idx + capacity].sum / tree[kRootIdx].min), -beta[0]);
  }
}

__global__ void SumTreeUpdateKernel(SumTree *tree, size_t capacity, float alpha, float *max_priority, size_t *indices,
                                    float *priorities, size_t batch_size) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < batch_size; i += gridDim.x * blockDim.x) {
    size_t idx = indices[i];
    float priority = powf(priorities[i], alpha);
    MsAtomicMax(max_priority, priority);

    idx += capacity;
    SumTreeInsert(tree, idx, priority);
  }
}

void SumTreeInit(SumTree *tree, float *max_priority, const size_t &capacity, hipStream_t stream) {
  size_t size = capacity * kNumSubNode;
  size_t block = std::min(size, kMaxThreadPerBlock);
  size_t grid = (size + block - 1) / block;
  SumTreeInitKernel<<<grid, block, 0, stream>>>(tree, max_priority, size);
}

void InitRandState(const size_t &batch_size, const uint64_t &seed, hiprandState *state, hipStream_t stream) {
  size_t block = std::min(batch_size, kMaxThreadPerBlock);
  size_t grid = (batch_size + block - 1) / block;
  InitRandStateKernel<<<grid, block, 0, stream>>>(seed, state);
}

void SumTreePush(SumTree *tree, const float &alpha, const size_t &idx, const size_t &capacity, float *priority,
                 float *max_priority, hipStream_t stream) {
  size_t idx_in_tree = idx + capacity;
  SumTreePushKernel<<<1, 1, 0, stream>>>(tree, alpha, idx_in_tree, priority, max_priority);
}

void SumTreeSample(SumTree *tree, hiprandState *state, const size_t &capacity, float *beta, const size_t &batch_size,
                   size_t *indices, float *weights, hipStream_t stream) {
  size_t block = std::min(batch_size, kMaxThreadPerBlock);
  size_t grid = (batch_size + block - 1) / block;
  SumTreeSampleKernel<<<grid, block, 0, stream>>>(tree, state, capacity, beta, batch_size, indices, weights);
}

void SumTreeUpdate(SumTree *tree, const size_t &capacity, const float &alpha, float *max_priority, size_t *indices,
                   float *priorities, const size_t &batch_size, hipStream_t stream) {
  size_t block = std::min(batch_size, kMaxThreadPerBlock);
  size_t grid = (batch_size + block - 1) / block;
  SumTreeUpdateKernel<<<grid, block, 0, stream>>>(tree, capacity, alpha, max_priority, indices, priorities, batch_size);
}

__global__ void FifoSliceKernel(const uint8_t *input, const size_t *indice, uint8_t *output, size_t batch_size,
                                size_t column) {
  for (size_t num = blockIdx.x * blockDim.x + threadIdx.x; num < batch_size * column; num += blockDim.x * gridDim.x) {
    size_t i = num / column;
    size_t j = num % column;
    size_t read_idex = indice[i] * column + j;
    output[num] = input[read_idex];
  }
}

void FifoSlice(const uint8_t *input, const size_t *indice, uint8_t *output, size_t batch_size, size_t column,
               hipStream_t stream) {
  size_t num = batch_size * column;
  size_t block = std::min(num, kMaxThreadPerBlock);
  size_t grid = (num + block - 1) / block;
  FifoSliceKernel<<<grid, block, 0, stream>>>(input, indice, output, batch_size, column);
  return;
}
