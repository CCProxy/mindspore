#include "hip/hip_runtime.h"
/**
 * Copyright 2020-2021 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/relu_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/util.cuh"

template <typename T>
__global__ void ReluV2Kernel(const size_t num, const T *x, T *y, uint32_t *mask) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < num; i += blockDim.x * gridDim.x) {
    T v = x[i];
    bool p = v > static_cast<T>(0);
    y[i] = p ? v : static_cast<T>(0);

    auto warp_predict = BallotSync(p, __activemask());
    if (LaneId() == 0) {
      mask[WarpId(i)] = warp_predict;
    }
  }
}

template <typename T>
void ReluV2(const size_t num, const T *x, T *y, uint32_t *mask, hipStream_t hip_stream) {
  ReluV2Kernel<<<kBlocksPerGrid(num), kThreadsPerBlock, 0, hip_stream>>>(num, x, y, mask);
}

template <typename T>
__global__ void ReluGradV2Kernel(const size_t num, const T *dy, const uint32_t *mask, T *dx) {
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < num; i += blockDim.x * gridDim.x) {
    bool p = mask[WarpId(i)] & (1 << LaneId());
    dx[i] = p ? dy[i] : static_cast<T>(0);
  }
}

template <typename T>
void ReluGradV2(const size_t num, const T *dy, const uint32_t *mask, T *dx, hipStream_t hip_stream) {
  ReluGradV2Kernel<<<kBlocksPerGrid(num), kThreadsPerBlock, 0, hip_stream>>>(num, dy, mask, dx);
}

template CUDA_LIB_EXPORT void ReluV2(const size_t num, const double *x, double *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const float *x, float *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const half *x, half *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const int8_t *x, int8_t *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const int16_t *x, int16_t *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const int32_t *x, int32_t *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const int64_t *x, int64_t *y, uint32_t *mask,
                                     hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluV2(const size_t num, const uint8_t *x, uint8_t *y, uint32_t *mask,
                                     hipStream_t hip_stream);

template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const double *dy, const uint32_t *mask, double *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const float *dy, const uint32_t *mask, float *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const half *dy, const uint32_t *mask, half *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const int8_t *dy, const uint32_t *mask, int8_t *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const int16_t *dy, const uint32_t *mask, int16_t *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const int32_t *dy, const uint32_t *mask, int32_t *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const int64_t *dy, const uint32_t *mask, int64_t *dx,
                                         hipStream_t hip_stream);
template CUDA_LIB_EXPORT void ReluGradV2(const size_t num, const uint8_t *dy, const uint32_t *mask, uint8_t *dx,
                                         hipStream_t hip_stream);
