#include "hip/hip_runtime.h"
/**
 * Copyright 2022 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "adjustsaturation_impl.cuh"
#include <algorithm>
#include <cmath>

template <typename T>
__device__ __forceinline__ void rgb2hsv_cuda(const T cu_r, const T cu_g, const T cu_b, T *cu_h, T *cu_s, T *cu_v) {
  *cu_v = max(cu_r, max(cu_g, cu_b));
  const T cu_m = min(cu_r, min(cu_g, cu_b));
  const T cu_chroma = (*cu_v) - cu_m;
  if (cu_chroma > 0.0) {
    if ((*cu_v) == cu_r) {
      const T cu_num = (cu_g - cu_b) / cu_chroma;
      const T cu_sign = copysign(static_cast<T>(1), cu_num);
      *cu_h =
        ((cu_sign < 0.0) * 6.0 + cu_sign * fmodf(static_cast<float>(cu_sign * cu_num), static_cast<float>(6.0))) / 6.0;
    } else if ((*cu_v) == cu_g) {
      *cu_h = ((cu_b - cu_r) / cu_chroma + 2.0) / 6.0;
    } else {
      *cu_h = ((cu_r - cu_g) / cu_chroma + 4.0) / 6.0;
    }
  } else {
    *cu_h = 0.0;
  }
  if ((*cu_v) > 0.0) {
    *cu_s = cu_chroma / (*cu_v);
  } else {
    *cu_s = 0.0;
  }
  return;
}

template <typename T>
__device__ __forceinline__ void hsv2rgb_cuda(const T cu_h, const T cu_s, const T cu_v, T *cu_r, T *cu_g, T *cu_b) {
  const T cu_new_h = cu_h * 6.0;
  const T cu_chroma = cu_v * cu_s;
  const T cu_x = cu_chroma * (1.0f - abs(fmodf(static_cast<float>(cu_new_h), static_cast<float>(2.0)) - 1.0));
  const T cu_new_m = cu_v - cu_chroma;
  const bool cu_between_0_and_1 = cu_new_h >= 0.0f && cu_new_h < 1.0f;
  const bool cu_between_1_and_2 = cu_new_h >= 1.0f && cu_new_h < 2.0f;
  const bool cu_between_2_and_3 = cu_new_h >= 2.0f && cu_new_h < 3.0f;
  const bool cu_between_3_and_4 = cu_new_h >= 3.0f && cu_new_h < 4.0f;
  const bool cu_between_4_and_5 = cu_new_h >= 4.0f && cu_new_h < 5.0f;
  const bool cu_between_5_and_6 = cu_new_h >= 5.0f && cu_new_h < 6.0f;
  *cu_r = cu_chroma * (cu_between_0_and_1 || cu_between_5_and_6) + cu_x * (cu_between_1_and_2 || cu_between_4_and_5) +
         cu_new_m;
  *cu_g = cu_chroma * (cu_between_1_and_2 || cu_between_2_and_3) + cu_x * (cu_between_0_and_1 || cu_between_3_and_4) +
         cu_new_m;
  *cu_b = cu_chroma * (cu_between_3_and_4 || cu_between_4_and_5) + cu_x * (cu_between_2_and_3 || cu_between_5_and_6) +
         cu_new_m;
  return;
}

template <typename T>
__global__ void CalAdjustSaturationKernel(const size_t tuple_elements, const int channel_num, const T *cu_input,
                                          T *cu_output, const float *cu_saturation_scale) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < tuple_elements; idx += gridDim.x * blockDim.x) {
    T cu_new_h = 0;
    T cu_new_s = 0;
    T cu_new_v = 0;
    rgb2hsv_cuda(cu_input[channel_num * idx], cu_input[channel_num * idx + 1], cu_input[channel_num * idx + 2],
                 &cu_new_h, &cu_new_s, &cu_new_v);
    const float cu_scale = *cu_saturation_scale;
    cu_new_s = min(1.0f, max(0.0f, cu_new_s * cu_scale));
    hsv2rgb_cuda(cu_new_h, cu_new_s, cu_new_v, &cu_output[channel_num * idx], &cu_output[channel_num * idx + 1],
                 &cu_output[channel_num * idx + 2]);
  }
}

template <>
__global__ void CalAdjustSaturationKernel(const size_t tuple_elements, const int channel_num, const half *cu_input,
                                          half *cu_output, const float *cu_saturation_scale) {
  for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < tuple_elements; idx += gridDim.x * blockDim.x) {
    float cu_new_h = 0;
    float cu_new_s = 0;
    float cu_new_v = 0;
    rgb2hsv_cuda(__half2float(cu_input[channel_num * idx]), __half2float(cu_input[channel_num * idx + 1]),
                 __half2float(cu_input[channel_num * idx + 2]), &cu_new_h, &cu_new_s, &cu_new_v);
    const float cu_scale = *cu_saturation_scale;
    cu_new_s = min(1.0f, max(0.0f, cu_new_s * cu_scale));
    float cu_r = 0;
    float cu_g = 0;
    float cu_b = 0;
    hsv2rgb_cuda(cu_new_h, cu_new_s, cu_new_v, &cu_r, &cu_g, &cu_b);
    cu_output[channel_num * idx] = __float2half(cu_r);
    cu_output[channel_num * idx + 1] = __float2half(cu_g);
    cu_output[channel_num * idx + 2] = __float2half(cu_b);
  }
}

template <typename T>
void CalAdjustSaturation(const int input_elements, const T *input, T *output, const float *saturation_scale,
                         const uint32_t &device_id, hipStream_t hip_stream) {
  const int channel_num = 3;
  int tuple_element = input_elements / channel_num;
  int thread_num = tuple_element > 256 ? 256 : tuple_element;
  CalAdjustSaturationKernel<<<CUDA_BLOCKS_CAL(device_id, tuple_element, thread_num), thread_num, 0, hip_stream>>>(
    tuple_element, channel_num, input, output, saturation_scale);
}

template CUDA_LIB_EXPORT void CalAdjustSaturation<float>(const int input_elements, const float *input, float *output,
                                                         const float *saturation_scale, const uint32_t &device_id,
                                                         hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalAdjustSaturation<half>(const int input_elements, const half *input, half *output,
                                                        const float *saturation_scale, const uint32_t &device_id,
                                                        hipStream_t hip_stream);

template CUDA_LIB_EXPORT void CalAdjustSaturation<double>(const int input_elements, const double *input, double *output,
                                                          const float *saturation_scale, const uint32_t &device_id,
                                                          hipStream_t hip_stream);
