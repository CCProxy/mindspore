#include "hip/hip_runtime.h"
/**
 * Copyright 2020 Huawei Technologies Co., Ltd
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/layer_norm_grad_impl.cuh"
#include "plugin/device/gpu/kernel/cuda_impl/cuda_ops/layer_norm_impl.cuh"
#include "include/hip/hip_fp16.h"

constexpr int NUM_PER_THREAD_REDUCE = 4;
constexpr int WARP_SIZE = 32;

template <typename T>
inline __device__ T my_pow(T a, double b) {
  return pow(a, static_cast<float>(b));
}

template <>
inline __device__ half my_pow(half a, double b) {
  return __float2half(pow(__half2float(a), static_cast<float>(b)));
}

template <typename T>
inline __device__ void GammaAndBetaThreadReduce(const int &col, const int &row_dim, const int &col_dim,
                                                const int &mean_dim, const T &epsilon, const T *dy, const T *x,
                                                const T *mean, const T *var, T *dg, T *db) {
  int loop_num = (row_dim + NUM_PER_THREAD_REDUCE - 1) / NUM_PER_THREAD_REDUCE;
  for (int i = threadIdx.x; i < loop_num; i += blockDim.x) {
    for (int j = 0; j < NUM_PER_THREAD_REDUCE; j++) {
      int row = NUM_PER_THREAD_REDUCE * i + j;
      if (row >= row_dim) {
        return;
      }

      int pos = row * col_dim + col;
      int mean_offset = pos / mean_dim;
      dg[0] += dy[pos] * my_pow(var[mean_offset] + epsilon, -0.5) * (x[pos] - mean[mean_offset]);
      db[0] += dy[pos];
    }
  }
}

template <typename T>
inline __device__ void GammaAndBetaWarpReduce(T *dg, T *db) {
  for (int delta = (WARP_SIZE >> 1); delta > 0; delta >>= 1) {
    dg[0] += __shfl_down_sync(0xffffffff, dg[0], delta);
    db[0] += __shfl_down_sync(0xffffffff, db[0], delta);
  }
}

template <typename T>
inline __device__ void GammaAndBetaBlockReduce(const int &col, const int &row_dim, T *dg, T *db, T *dg_addr,
                                               T *db_addr) {
  // load data to share memory
  // thread(0, 32, 64, 96, ...) keep the data
  DynamicSharedMem<T> share_mem;
  if (threadIdx.x % WARP_SIZE == 0) {
    int offset = threadIdx.x / WARP_SIZE * 2;
    share_mem.addr()[offset] = dg[0];
    share_mem.addr()[offset + 1] = db[0];
  }
  __syncthreads();

  for (int stride = blockDim.x / WARP_SIZE / 2; stride > 0; stride >>= 1) {
    if (threadIdx.x < stride) {
      int offset = (threadIdx.x + stride) * 2;
      share_mem.addr()[threadIdx.x * 2] += share_mem.addr()[offset];
      share_mem.addr()[threadIdx.x * 2 + 1] += share_mem.addr()[offset + 1];
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    dg_addr[col] = share_mem.addr()[0];
    db_addr[col] = share_mem.addr()[1];
  }
}

template <typename T>
__global__ void GammaAndBetaPropKernel(const int row_dim, const int col_dim, const int mean_dim, const T epsilon,
                                       const T *dy, const T *x, const T *mean_addr, const T *var_addr, T *dg_addr,
                                       T *db_addr) {
  // row: [0:param_axis]
  // col: [param_axis:]
  // dg[i][j] = dy[i][j] * (var[i] + epsilon, -0.5) * (x[i][j] - mean[i])
  // dg[j] = \Sigma_{j}dg[i][j]
  for (int col = blockIdx.x; col < col_dim; col += gridDim.x) {
    T dg = 0;
    T db = 0;
    GammaAndBetaThreadReduce(col, row_dim, col_dim, mean_dim, epsilon, dy, x, mean_addr, var_addr, &dg, &db);
    GammaAndBetaWarpReduce(&dg, &db);
    GammaAndBetaBlockReduce(col, row_dim, &dg, &db, dg_addr, db_addr);
  }
}

constexpr int kTileSize = 8;
template <typename T>
struct alignas(sizeof(T) * kTileSize) TArray {
  T data[kTileSize];
};

template <typename T>
inline __device__ void TiledGammaAndBetaThreadReduce(const int &col, const int &row_dim, const int &col_dim,
                                                     const int &mean_dim, const T &epsilon, const T *dy, const T *x,
                                                     const T *mean, const T *var, T *dg, T *db) {
  for (int i = 0; i < kTileSize; ++i) {
    dg[i] = 0;
    db[i] = 0;
  }
  for (int i = threadIdx.x; i < row_dim; i += blockDim.x) {
    T dy_tile[kTileSize];
    T x_tile[kTileSize];
    TArray<T> *dy_tmp = reinterpret_cast<TArray<T> *>(&dy_tile);
    *dy_tmp = *reinterpret_cast<const TArray<T> *>(&dy[i * col_dim + col]);
    TArray<T> *x_tmp = reinterpret_cast<TArray<T> *>(x_tile);
    *x_tmp = *reinterpret_cast<const TArray<T> *>(&x[i * col_dim + col]);
    for (int j = 0; j < kTileSize; ++j) {
      dg[j] += dy_tile[j] * my_pow(var[i] + epsilon, -0.5) * (x_tile[j] - mean[i]);
      db[j] += dy_tile[j];
    }
  }
}

template <typename T>
inline __device__ void TiledGammaAndBetaWarpReduce(T *dg, T *db) {
  for (int i = 0; i < kTileSize; ++i) {
    for (int delta = (WARP_SIZE >> 1); delta > 0; delta >>= 1) {
      dg[i] += __shfl_down_sync(0xffffffff, dg[i], delta);
      db[i] += __shfl_down_sync(0xffffffff, db[i], delta);
    }
  }
}

template <typename T>
inline __device__ void TiledGammaAndBetaBlockReduce(const int &col, const int &row_dim, T *dg, T *db, T *dg_addr,
                                                    T *db_addr) {
  DynamicSharedMem<T> share_mem;
  if (threadIdx.x % WARP_SIZE == 0) {
    int offset = threadIdx.x / WARP_SIZE * 2 * kTileSize;
    for (int i = 0; i < kTileSize; ++i) {
      share_mem.addr()[offset + i * 2] = dg[i];
      share_mem.addr()[offset + i * 2 + 1] = db[i];
    }
  }
  __syncthreads();

  for (int stride = blockDim.x / WARP_SIZE / 2; stride > 0; stride >>= 1) {
    if (threadIdx.x < stride) {
      int offset = (threadIdx.x + stride) * 2 * kTileSize;
      for (int i = 0; i < kTileSize; ++i) {
        share_mem.addr()[threadIdx.x * 2 * kTileSize + 2 * i] += share_mem.addr()[offset + 2 * i];
        share_mem.addr()[threadIdx.x * 2 * kTileSize + 2 * i + 1] += share_mem.addr()[offset + 2 * i + 1];
      }
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    for (int i = 0; i < kTileSize; ++i) {
      dg_addr[col + i] = share_mem.addr()[2 * i];
      db_addr[col + i] = share_mem.addr()[2 * i + 1];
    }
  }
}

template <typename T>
__global__ void TiledGammaAndBetaPropKernel(const int row_dim, const int col_dim, const int mean_dim, const T epsilon,
                                            const T *dy, const T *x, const T *mean_addr, const T *var_addr, T *dg_addr,
                                            T *db_addr) {
  for (int col = blockIdx.x * kTileSize; col < col_dim; col += gridDim.x * kTileSize) {
    T dg[kTileSize];
    T db[kTileSize];
    TiledGammaAndBetaThreadReduce(col, row_dim, col_dim, mean_dim, epsilon, dy, x, mean_addr, var_addr, dg, db);
    TiledGammaAndBetaWarpReduce(dg, db);
    TiledGammaAndBetaBlockReduce(col, row_dim, dg, db, dg_addr, db_addr);
  }
}

template <typename T>
inline __device__ void InputThreadReduce(const int &row, const int &col_dim, const int &param_dim, const T &epsilon,
                                         T *sum1, T *sum2, T *sum3, const T *dy, const T *x, const T *mean,
                                         const T *var, const T *gamma) {
  int loop_num = (col_dim + NUM_PER_THREAD_REDUCE - 1) / NUM_PER_THREAD_REDUCE;
  for (int i = threadIdx.x; i < loop_num; i += blockDim.x) {
    for (int j = 0; j < NUM_PER_THREAD_REDUCE; j++) {
      int col = NUM_PER_THREAD_REDUCE * i + j;
      if (col >= col_dim) {
        sum1[0] = -0.5 * sum1[0] * my_pow(var[row] + epsilon, -1.5);
        sum3[0] = -2.0 * sum3[0];
        return;
      }

      int pos = row * col_dim + col;
      int gamma_offset = pos % param_dim;
      T v1 = dy[pos] * gamma[gamma_offset];
      T v2 = x[pos] - mean[row];

      sum1[0] += v1 * v2;
      sum2[0] += v1;
      sum3[0] += v2;
    }
  }
  sum1[0] = -0.5 * sum1[0] * my_pow(var[row] + epsilon, -1.5);
  sum3[0] = -2.0 * sum3[0];
}

template <>
inline __device__ void InputThreadReduce(const int &row, const int &col_dim, const int &param_dim, const half &epsilon,
                                         half *sum1, half *sum2, half *sum3, const half *dy, const half *x,
                                         const half *mean, const half *var, const half *gamma) {
  int loop_num = (col_dim + NUM_PER_THREAD_REDUCE - 1) / NUM_PER_THREAD_REDUCE;
  for (int i = threadIdx.x; i < loop_num; i += blockDim.x) {
    for (int j = 0; j < NUM_PER_THREAD_REDUCE; j++) {
      int col = NUM_PER_THREAD_REDUCE * i + j;
      if (col >= col_dim) {
        sum1[0] = __float2half(-0.5) * sum1[0] * my_pow(var[row] + epsilon, -1.5);
        sum3[0] = __float2half(-2.0) * sum3[0];
        return;
      }

      int pos = row * col_dim + col;
      int gamma_offset = pos % param_dim;
      half v1 = dy[pos] * gamma[gamma_offset];
      half v2 = x[pos] - mean[row];

      sum1[0] += v1 * v2;
      sum2[0] += v1;
      sum3[0] += v2;
    }
  }
  sum1[0] = __float2half(-0.5) * sum1[0] * my_pow(var[row] + epsilon, -1.5);
  sum3[0] = __float2half(-2.0) * sum3[0];
}

template <typename T>
inline __device__ void TiledInputThreadReduce(const int &row, const int &col_dim, const int &param_dim,
                                              const T &epsilon, T *sum1, T *sum2, T *sum3, const T *dy, const T *x,
                                              const T *mean, const T *var, const T *gamma) {
  for (int i = threadIdx.x * kTileSize; i < col_dim; i += blockDim.x * kTileSize) {
    int pos = row * col_dim + i;
    T dy_tile[kTileSize];
    T x_tile[kTileSize];
    TArray<T> *dy_tmp = reinterpret_cast<TArray<T> *>(&dy_tile);
    *dy_tmp = *reinterpret_cast<const TArray<T> *>(&dy[pos]);
    TArray<T> *x_tmp = reinterpret_cast<TArray<T> *>(x_tile);
    *x_tmp = *reinterpret_cast<const TArray<T> *>(&x[pos]);

    for (int j = 0; j < kTileSize; ++j) {
      T v1 = dy_tile[j] * gamma[i + j];
      T v2 = x_tile[j] - mean[row];
      sum1[0] += v1 * v2;
      sum2[0] += v1;
      sum3[0] += v2;
    }
  }
  sum1[0] = (T)(-0.5) * sum1[0] * my_pow(var[row] + epsilon, -1.5);
  sum3[0] = (T)(-2.0) * sum3[0];
}

template <typename T>
inline __device__ void InputWarpReduce(T *sum1, T *sum2, T *sum3) {
  for (int delta = (WARP_SIZE >> 1); delta > 0; delta >>= 1) {
    sum1[0] += __shfl_down_sync(0xffffffff, sum1[0], delta);
    sum2[0] += __shfl_down_sync(0xffffffff, sum2[0], delta);
    sum3[0] += __shfl_down_sync(0xffffffff, sum3[0], delta);
  }
}

template <typename T>
inline __device__ void InputBlockReduce(const int &col_dim, T *sum1, T *sum2, T *sum3, T *share_mem) {
  // load data to share memory
  // thread(0, 32, 64, 96, ...) keep the data
  if (threadIdx.x % WARP_SIZE == 0) {
    int offset = threadIdx.x / WARP_SIZE * 3;
    share_mem[offset] = sum1[0];
    share_mem[offset + 1] = sum2[0];
    share_mem[offset + 2] = sum3[0];
  }
  __syncthreads();

  for (int stride = blockDim.x / WARP_SIZE / 2; stride > 0; stride >>= 1) {
    if (threadIdx.x < stride) {
      int offset = (threadIdx.x + stride) * 3;
      share_mem[threadIdx.x * 3] += share_mem[offset];
      share_mem[threadIdx.x * 3 + 1] += share_mem[offset + 1];
      share_mem[threadIdx.x * 3 + 2] += share_mem[offset + 2];
    }
  }
  __syncthreads();
}

template <typename T>
inline __device__ void InputProp(const int &row, const int &col_dim, const int &param_dim, const T &epsilon,
                                 const T *dy, const T *x, const T *mean, const T *var, const T *gamma, T *dx,
                                 const T *share_mem) {
  T v3 = my_pow(var[row] + epsilon, -0.5);
  T v4 = share_mem[0] * (2.0 / col_dim);
  T v5 = (-1.0 * v3 * share_mem[1] + (1.0 / col_dim) * share_mem[0] * share_mem[2]) * (1.0 / col_dim);
  for (int col = threadIdx.x; col < col_dim; col += blockDim.x) {
    int pos = (row * col_dim + col);
    int gamma_offset = pos % param_dim;
    T v1 = dy[pos] * gamma[gamma_offset];
    T v2 = x[pos] - mean[row];
    dx[pos] = v1 * v3 + v4 * v2 + v5;
  }
}

template <>
inline __device__ void InputProp(const int &row, const int &col_dim, const int &param_dim, const half &epsilon,
                                 const half *dy, const half *x, const half *mean, const half *var, const half *gamma,
                                 half *dx, const half *share_mem) {
  half v3 = my_pow(var[row] + epsilon, -0.5);
  half v4 = share_mem[0] * __float2half(2.0 / col_dim);
  half v5 = (__float2half(-1.0) * v3 * share_mem[1] + __float2half(1.0 / col_dim) * share_mem[0] * share_mem[2]) *
            __float2half(1.0 / col_dim);
  for (int col = threadIdx.x; col < col_dim; col += blockDim.x) {
    int pos = (row * col_dim + col);
    int gamma_offset = pos % param_dim;
    half v1 = dy[pos] * gamma[gamma_offset];
    half v2 = x[pos] - mean[row];
    dx[pos] = v1 * v3 + v4 * v2 + v5;
  }
}

template <typename T>
__global__ void InputPropKernel(const int row_dim, const int col_dim, const int param_dim, const T epsilon, const T *dy,
                                const T *x, const T *mean, const T *var, const T *gamma, T *dx) {
  for (int row = blockIdx.x; row < row_dim; row += gridDim.x) {
    T sum1 = 0;
    T sum2 = 0;
    T sum3 = 0;
    DynamicSharedMem<T> share_mem;
    InputThreadReduce(row, col_dim, param_dim, epsilon, &sum1, &sum2, &sum3, dy, x, mean, var, gamma);
    InputWarpReduce(&sum1, &sum2, &sum3);
    InputBlockReduce(col_dim, &sum1, &sum2, &sum3, share_mem.addr());
    InputProp(row, col_dim, param_dim, epsilon, dy, x, mean, var, gamma, dx, share_mem.addr());
  }
}

template <typename T>
inline __device__ void TiledInputProp(const int &row, const int &col_dim, const int &param_dim, const T &epsilon,
                                      const T *dy, const T *x, const T *mean, const T *var, const T *gamma, T *dx,
                                      const T *share_mem) {
  T col_inv = (T)(1.0 / col_dim);
  T v3 = my_pow(var[row] + epsilon, -0.5);
  T v4 = share_mem[0] * col_inv * (T)(2.0);
  T v5 = (col_inv * share_mem[0] * share_mem[2] - v3 * share_mem[1]) * col_inv;
  for (int col = threadIdx.x * kTileSize; col < col_dim; col += blockDim.x * kTileSize) {
    int pos = row * col_dim + col;
    T dy_tile[kTileSize];
    T x_tile[kTileSize];
    T dx_tile[kTileSize];
    TArray<T> *dy_tmp = reinterpret_cast<TArray<T> *>(&dy_tile);
    *dy_tmp = *reinterpret_cast<const TArray<T> *>(&dy[pos]);
    TArray<T> *x_tmp = reinterpret_cast<TArray<T> *>(x_tile);
    *x_tmp = *reinterpret_cast<const TArray<T> *>(&x[pos]);

    for (int j = 0; j < kTileSize; ++j) {
      T v1 = dy_tile[j] * gamma[col + j];
      T v2 = x_tile[j] - mean[row];
      dx_tile[j] = v1 * v3 + v4 * v2 + v5;
    }
    TArray<T> *dx_tmp = reinterpret_cast<TArray<T> *>(&dx[pos]);
    *dx_tmp = *reinterpret_cast<TArray<T> *>(dx_tile);
  }
}

template <typename T>
__global__ void TiledInputPropKernel(const int row_dim, const int col_dim, const int param_dim, const T epsilon,
                                     const T *dy, const T *x, const T *mean, const T *var, const T *gamma, T *dx) {
  for (int row = blockIdx.x; row < row_dim; row += gridDim.x) {
    T sum1 = 0;
    T sum2 = 0;
    T sum3 = 0;
    DynamicSharedMem<T> share_mem;
    TiledInputThreadReduce(row, col_dim, param_dim, epsilon, &sum1, &sum2, &sum3, dy, x, mean, var, gamma);
    InputWarpReduce(&sum1, &sum2, &sum3);
    InputBlockReduce(col_dim, &sum1, &sum2, &sum3, share_mem.addr());
    TiledInputProp(row, col_dim, param_dim, epsilon, dy, x, mean, var, gamma, dx, share_mem.addr());
  }
}

template <typename T>
void LayerNormGrad(const int &row_dim, const int &col_dim, const int &param_dim, const T &epsilon, const T *dy,
                   const T *x, const T *mean, const T *var, const T *gamma, T *dx, T *dg, T *db, hipStream_t stream) {
  const int thread_per_block = 256;
  int share_mem_size = thread_per_block / WARP_SIZE * 3 * sizeof(T);

  int param_reduce_dim = row_dim * col_dim / param_dim;
  int grid_size = param_dim;
  if (col_dim == param_dim && grid_size % kTileSize == 0 && col_dim % kTileSize == 0) {
    TiledInputPropKernel<<<row_dim, thread_per_block, share_mem_size, stream>>>(row_dim, col_dim, param_dim, epsilon,
                                                                                dy, x, mean, var, gamma, dx);
    share_mem_size = thread_per_block / WARP_SIZE * 2 * kTileSize * sizeof(T);
    grid_size /= kTileSize;
    TiledGammaAndBetaPropKernel<<<grid_size, thread_per_block, share_mem_size, stream>>>(
      param_reduce_dim, param_dim, col_dim, epsilon, dy, x, mean, var, dg, db);
  } else {
    InputPropKernel<<<row_dim, thread_per_block, share_mem_size, stream>>>(row_dim, col_dim, param_dim, epsilon, dy, x,
                                                                           mean, var, gamma, dx);
    share_mem_size = thread_per_block / WARP_SIZE * 2 * sizeof(T);
    GammaAndBetaPropKernel<<<grid_size, thread_per_block, share_mem_size, stream>>>(
      param_reduce_dim, param_dim, col_dim, epsilon, dy, x, mean, var, dg, db);
  }
}

template CUDA_LIB_EXPORT void LayerNormGrad(const int &row_dim, const int &col_dim, const int &param_dim,
                                            const float &epsilon, const float *dy, const float *x, const float *mean,
                                            const float *var, const float *gamma, float *dx, float *dg, float *db,
                                            hipStream_t stream);
template CUDA_LIB_EXPORT void LayerNormGrad(const int &row_dim, const int &col_dim, const int &param_dim,
                                            const half &epsilon, const half *dy, const half *x, const half *mean,
                                            const half *var, const half *gamma, half *dx, half *dg, half *db,
                                            hipStream_t stream);
